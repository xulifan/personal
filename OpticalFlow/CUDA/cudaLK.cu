#include "hip/hip_runtime.h"
#include "cudaLK.h"
#include <stdio.h>

const float scaling[] = {1, 0.5f, 0.25f, 0.125f, 0.0625f, 0.03125f, 0.015625f, 0.0078125f};

// Can't use an array of texture<> !! so we'll just re-use the one texture buffer for each image
texture<float, 2, hipReadModeElementType> texRef_pyramid_prev;
texture<float, 2, hipReadModeElementType> texRef_pyramid_cur;

__global__ void convertToGrey(unsigned char *d_in, float *d_out, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < N) 
        d_out[idx] = d_in[idx*3]*0.1144f + d_in[idx*3+1]*0.5867f + d_in[idx*3+2]*0.2989f;
}

__global__ void pyrDownsample(float *in, int w1, int h1, float *out, int w2, int h2)
{
    // Input has to be greyscale
    int x2 = blockIdx.x*blockDim.x + threadIdx.x;
    int y2 = blockIdx.y*blockDim.y + threadIdx.y;

    if( (x2 < w2) && (y2 < h2) ) {    
        int x = x2*2;
        int y = y2*2;
        int x_1 = x-1;
        int y_1 = y-1;
        int x_2 = x+1;
        int y_2 = y+1;

        if(x_1 < 0) x_1 = 0;
        if(y_1 < 0) y_1 = 0;
        if(x_2 >= w1) x_2 = w1 - 1;
        if(y_2 >= h1) y_2 = h1 - 1;

        out[y2*w2 + x2] = 0.25f*in[y*w1+x] + 0.125f*(in[y*w1+x_1] + in[y*w1+x_2] + in[y_1*w1+x] + in[y_2*w1+x]) +
                          0.0625f*(in[y_1*w1+x_1] + in[y_2*w1+x_1] + in[y_1*w1+x_2] + in[y_2*w1+x_2]);
    }
}
    
    
__global__ void smoothX(float *in, int w, int h, float *out) 
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= w || y >= h)
        return;

    int idx = y*w;

    int a = x-2;
    int b = x-1;
    int c = x;
    int d = x+1;
    int e = x+2;

    if(a < 0) a = 0;
    if(b < 0) b = 0;
    if(c >= w) c = w-1;
    if(d >= w) d = w-1;

    out[y*w+x] = 0.0625f*in[idx+a] + 0.25f*in[idx+b] + 0.375f*in[idx+c] + 0.25f*in[idx+d] + 0.0625f*in[idx+e];
}
    
__global__ void smoothY(float *in, int w, int h, float *out) 
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= w || y >= h)
        return;

    int a = y-2;
    int b = y-1;
    int c = y;
    int d = y+1;
    int e = y+2;

    if(a < 0) a = 0;
    if(b < 0) b = 0;
    if(c >= h) c = h-1;
    if(d >= h) d = h-1;

    out[y*w+x] = 0.0625f*in[a*w+x] + 0.25f*in[b*w+x] + 0.375f*in[c*w+x] + 0.25f*in[d*w+x] + 0.0625f*in[e*w+x];
}

// Call recursively
// w/h - original dimension of image

__global__ void track(const int w, const int h, 
                      const int pyr_w, const int pyr_h, 
                      float scaling, int level, char initGuess, 
                      float *dx, float *dy, char *status)
{        
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = y*w + x;
    
    if(x > w-1 || y > h-1) 
        return;

    if(status[idx] == 0)
        return;

    float prev_x = x*scaling;
    float prev_y = y*scaling;

    float Vx, Vy;
    float cur_x, cur_y;
    float sum_Ixx = 0;
    float sum_Ixy = 0;
    float sum_Iyy = 0;
    float sum_Ixt;
    float sum_Iyt;
    float Ix, Iy, It;
    int xx, yy;
    float det, D;
    float I, J;
    float vx, vy;
    int j;

    if(initGuess) {
        Vx = 0;
        Vy = 0;
        cur_x = prev_x;
        cur_y = prev_y;
    }
    else {
        Vx = dx[idx];
        Vy = dy[idx];
        cur_x = prev_x + Vx;
        cur_y = prev_y + Vy;
    }
  
    // Calculate spatial gradient 
    for(yy=-PATCH_R; yy <= PATCH_R; yy++) {
        for(xx=-PATCH_R; xx <= PATCH_R; xx++) {  
            Ix = (tex2D(texRef_pyramid_prev, prev_x + xx+1, prev_y + yy) - tex2D(texRef_pyramid_prev, prev_x + xx-1, prev_y + yy))*0.5f;
            Iy = (tex2D(texRef_pyramid_prev, prev_x + xx, prev_y + yy+1) - tex2D(texRef_pyramid_prev, prev_x + xx, prev_y + yy-1))*0.5f;

            sum_Ixx += Ix*Ix;
            sum_Ixy += Ix*Iy;
            sum_Iyy += Iy*Iy;
        }
    }

    det = sum_Ixx*sum_Iyy - sum_Ixy*sum_Ixy;

    if(det < 0.00001f) {
        status[idx] = 0;
        return;
    }

    D = 1/det;

    // Iteration part
    for(j=0; j < 10; j++) {
        if(cur_x < 0 || cur_x > pyr_w || cur_y < 0 || cur_y > pyr_h) {
            status[idx] = 0;
            return;
        }

        sum_Ixt = 0;
        sum_Iyt = 0;

        // No explicit handling of pixels outside the image ... maybe we don't have to because the hardware interpolation scheme
        // will always give a result for pixels outside the image. How greatly the duplicated pixel values affect the result is unknown at the moment.
        for(yy=-PATCH_R; yy <= PATCH_R; yy++) {
            for(xx=-PATCH_R; xx <= PATCH_R; xx++) {            
                I = tex2D(texRef_pyramid_prev, prev_x + xx, prev_y + yy);   
                J = tex2D(texRef_pyramid_cur, cur_x + xx, cur_y + yy);

                Ix = (tex2D(texRef_pyramid_prev, prev_x + xx+1, prev_y + yy) - tex2D(texRef_pyramid_prev, prev_x + xx-1, prev_y + yy))*0.5f;
                Iy = (tex2D(texRef_pyramid_prev, prev_x + xx, prev_y + yy+1) - tex2D(texRef_pyramid_prev, prev_x + xx, prev_y + yy-1))*0.5f;

                It = J - I;

                sum_Ixt += Ix*It;
                sum_Iyt += Iy*It;
            }            
        }

        // Find the inverse of the 2x2 matrix using a mix of determinant and adjugate matrix
        // http://cnx.org/content/m19446/latest/
        vx = D*(-sum_Iyy*sum_Ixt + sum_Ixy*sum_Iyt);
        vy = D*( sum_Ixy*sum_Ixt - sum_Ixx*sum_Iyt);

        Vx += vx;
        Vy += vy;
        cur_x += vx;
        cur_y += vy;
 
        // Movement very small
        if(fabsf(vx) < 0.01f && fabsf(vy) < 0.01f)
            break;
    }

    if(level != 0) {
        cur_x += cur_x;
        cur_y += cur_y;

        Vx += Vx;
        Vy += Vy;
    }

    dx[idx] = Vx;
    dy[idx] = Vy;
}

cudaLK::cudaLK()
{

}

cudaLK::~cudaLK()
{
    for(int i=0; i < LEVELS; i++) {
        hipFree(gpu_img_pyramid_prev[i]);
        hipFree(gpu_img_pyramid_cur[i]);
    }

    hipFree(gpu_smoothed_prev_x);
    hipFree(gpu_smoothed_cur_x);
    hipFree(gpu_smoothed_prev);
    hipFree(gpu_smoothed_cur);

    hipFreeArray(gpu_array_pyramid_prev);
    hipFreeArray(gpu_array_pyramid_prev_Ix);
    hipFreeArray(gpu_array_pyramid_prev_Iy);
    hipFreeArray(gpu_array_pyramid_cur);

    hipFree(gpu_dx);
    hipFree(gpu_dy);
    hipFree(gpu_status);

    delete [] dx;
    delete [] dy;
    delete [] status;
}

void cudaLK::checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 

void cudaLK::initMem()
{
    hipMalloc((void**)&gpu_img_prev_RGB, sizeof(char)*w*h*3);
    hipMalloc((void**)&gpu_img_cur_RGB, sizeof(char)*w*h*3);
    hipMalloc((void**)&gpu_img_pyramid_prev[0], sizeof(float)*w*h);
    hipMalloc((void**)&gpu_img_pyramid_cur[0], sizeof(float)*w*h);

    hipMalloc((void**)&gpu_smoothed_prev_x, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_smoothed_cur_x, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_smoothed_prev, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_smoothed_cur, sizeof(float)*w*h);

    // Texture
    hipMallocArray(&gpu_array_pyramid_prev, &texRef_pyramid_prev.channelDesc, w, h);
    hipMallocArray(&gpu_array_pyramid_cur, &texRef_pyramid_cur.channelDesc, w, h);
    hipBindTextureToArray(texRef_pyramid_prev, gpu_array_pyramid_prev);
    hipBindTextureToArray(texRef_pyramid_cur, gpu_array_pyramid_cur);

    texRef_pyramid_prev.normalized = 0;
    texRef_pyramid_prev.filterMode = hipFilterModeLinear;
    texRef_pyramid_prev.addressMode[0] = hipAddressModeClamp;
    texRef_pyramid_prev.addressMode[1] = hipAddressModeClamp;

    texRef_pyramid_cur.normalized = 0;
    texRef_pyramid_cur.filterMode = hipFilterModeLinear;
    texRef_pyramid_cur.addressMode[0] = hipAddressModeClamp;
    texRef_pyramid_cur.addressMode[1] = hipAddressModeClamp;

    hipMalloc((void**)&gpu_dx, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_dy, sizeof(float)*w*h);
    hipMalloc((void**)&gpu_status, sizeof(char)*w*h);

    int _w = w;
    int _h = h;

    dx = new float[w*h];
    dy = new float[w*h];
    status = new char[w*h];

    pyr_w[0] = w;
    pyr_h[0] = h;

    for(int i=1; i < LEVELS; i++) {
        _w /= 2;
        _h /= 2;
        pyr_w[i] = _w;
        pyr_h[i] = _h;

        hipMalloc((void**)&gpu_img_pyramid_prev[i], sizeof(float)*_w*_h);
        hipMalloc((void**)&gpu_img_pyramid_cur[i], sizeof(float)*_w*_h);
    }
}

void cudaLK::run(unsigned char *prev, unsigned char *cur, int _w, int _h)
{;
    w = _w;
    h = _h;
    initMem();

    int nThreadsX = NTHREAD_X;
    int nThreadsY = NTHREAD_Y;

    int blocksW = w/nThreadsX + ((w % nThreadsX)?1:0);
    int blocksH = h/nThreadsY + ((h % nThreadsY )?1:0);
    dim3 blocks(blocksW, blocksH);
    dim3 threads(nThreadsX, nThreadsY);
    int blocks1D = (w*h)/256 + (w*h % 256?1:0); // for greyscale
  
    int start = getTimeNow();
    int s;

    // Copy image to GPU 
    s = getTimeNow();
    hipMemcpy(gpu_img_prev_RGB, prev, w*h*3, hipMemcpyHostToDevice);  
    hipMemcpy(gpu_img_cur_RGB, cur, w*h*3, hipMemcpyHostToDevice);  
    checkCUDAError("start");

    printf("Copying 2 images from CPU to GPU: %d ms\n", getTimeNow() - s);

    // RGB -> grey
    s = getTimeNow();
    convertToGrey<<<blocks1D, 256>>>(gpu_img_prev_RGB, gpu_img_pyramid_prev[0], w*h);
    convertToGrey<<<blocks1D, 256>>>(gpu_img_cur_RGB, gpu_img_pyramid_cur[0], w*h);
    hipDeviceSynchronize();
    checkCUDAError("convertToGrey");
    printf("Converting from RGB to greyscale: %d ms\n", getTimeNow() - s);

  
    s = getTimeNow();

    for(int i=0; i < LEVELS-1; i++) {
        smoothX<<<blocks, threads>>>(gpu_img_pyramid_prev[i], pyr_w[i], pyr_h[i], gpu_smoothed_prev_x);
        smoothX<<<blocks, threads>>>(gpu_img_pyramid_cur[i], pyr_w[i], pyr_h[i], gpu_smoothed_cur_x);
        hipDeviceSynchronize();
        smoothY<<<blocks, threads>>>(gpu_smoothed_prev_x, pyr_w[i], pyr_h[i], gpu_smoothed_prev);
        smoothY<<<blocks, threads>>>(gpu_smoothed_cur_x, pyr_w[i], pyr_h[i], gpu_smoothed_cur);
        hipDeviceSynchronize();

        pyrDownsample<<<blocks, threads>>>(gpu_smoothed_prev, pyr_w[i], pyr_h[i], gpu_img_pyramid_prev[i+1], pyr_w[i+1], pyr_h[i+1]);
        pyrDownsample<<<blocks, threads>>>(gpu_smoothed_cur,  pyr_w[i], pyr_h[i], gpu_img_pyramid_cur[i+1],  pyr_w[i+1], pyr_h[i+1]);
        hipDeviceSynchronize();

        checkCUDAError("pyrDownsample here");  
    }

    printf("Generating the pyramids: %d ms\n", getTimeNow() - s);

    s = getTimeNow();
    hipMemset(gpu_status, 1, sizeof(char)*w*h);

    // Do the actual tracking
    for(int l=LEVELS-1; l >= 0; l--) {

        hipMemcpy2DToArray(gpu_array_pyramid_prev, 0, 0, gpu_img_pyramid_prev[l], 
                            sizeof(float)*pyr_w[l], sizeof(float)*pyr_w[l], pyr_h[l], hipMemcpyDeviceToDevice);

        hipMemcpy2DToArray(gpu_array_pyramid_cur, 0, 0, gpu_img_pyramid_cur[l], 
                            sizeof(float)*pyr_w[l], sizeof(float)*pyr_w[l], pyr_h[l], hipMemcpyDeviceToDevice);

        track<<<blocks, threads>>>(w, h, pyr_w[l], pyr_w[l], scaling[l], l, (l == LEVELS-1), gpu_dx, gpu_dy, gpu_status);

        hipDeviceSynchronize();
    }

    printf("Tracking: %d ms\n", getTimeNow() - s);

    // Copy back results 
    s = getTimeNow();
    hipMemcpy(dx, gpu_dx, sizeof(float)*w*h, hipMemcpyDeviceToHost);  
    hipMemcpy(dy, gpu_dy, sizeof(float)*w*h, hipMemcpyDeviceToHost);  
    hipMemcpy(status, gpu_status, sizeof(char)*w*h, hipMemcpyDeviceToHost);  
    printf("Copying results from GPU to CPU: %d ms\n", getTimeNow() - s);

    printf("Total time for cudaLK: %d ms\n", getTimeNow() - start);
}

